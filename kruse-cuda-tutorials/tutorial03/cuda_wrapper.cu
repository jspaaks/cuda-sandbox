#include <hip/hip_runtime.h>

#include <stdio.h>

__global__ void processKernel(int *numberArray, int N)
{
  int idx = blockIdx.x*blockDim.x + threadIdx.x;

  if (idx<N)
    numberArray[idx] = numberArray[idx] + 1;
}

extern void cuda_doStuff(int *array_in, int *array_out, int N)
{
  int *numbers_d;
					  
  int numberOfBlocks = 1;	      				
  int threadsPerBlock = N;			
  int maxNumberOfThreads = N;

  hipMalloc((void **) &numbers_d, sizeof(int)*N);

  hipMemcpy(numbers_d, array_in, sizeof(int)*N, hipMemcpyHostToDevice);
  processKernel<<<numberOfBlocks, threadsPerBlock>>>(numbers_d, maxNumberOfThreads);
  hipDeviceSynchronize();
  hipMemcpy(array_out, numbers_d, sizeof(int)*N, hipMemcpyDeviceToHost);

  hipFree(numbers_d);

  return;
}
